#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <map>
#include <vector>

#include <thrust/adjacent_difference.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>
#include <thrust/sort.h>

#include "simulation.h"

void cuda_check(hipError_t e, const char* file, int line)
{
	if (e != hipSuccess)
	{
		std::printf("CUDA API failed at %s:%d with error: %s (%d)\n", file, line, hipGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

#define CUDA_CHECK(func) cuda_check(func, __FILE__, __LINE__)

void statistics_windows_probs(std::vector<std::map<size_t, float>>& probs, float window_size, float max_time,
							  thrust::device_ptr<size_t> traj_states, thrust::device_ptr<float> traj_times,
							  size_t max_traj_len, size_t n_trajectories)
{
	thrust::device_vector<float> diffs(n_trajectories * max_traj_len);

	thrust::adjacent_difference(traj_times, traj_times + n_trajectories * max_traj_len, diffs.begin());
	std::cout << "adj diff" << std::endl;

	auto begin = thrust::make_zip_iterator(traj_states, traj_times, diffs.begin());
	auto end = thrust::make_zip_iterator(traj_states + n_trajectories * max_traj_len,
										 traj_times + n_trajectories * max_traj_len, diffs.end());

	size_t window_idx = 0;

	for (float cumul_time = window_size; cumul_time < max_time; cumul_time += window_size, window_idx++)
	{
		float w_b = cumul_time - window_size;
		float w_e = cumul_time;

		// find states in the window by moving them to the front
		auto partition_point =
			thrust::partition(begin, end, [w_b, w_e] __device__(const thrust::tuple<size_t, float, float>& t) {
				const float b = thrust::get<1>(t) - thrust::get<2>(t);
				const float e = thrust::get<1>(t);

				return !(e < w_b || b >= w_e);
			});

		
		std::cout << "part" << std::endl;

		size_t states_in_window_size = partition_point - begin;

		if (states_in_window_size == 0)
			continue;

		thrust::sort_by_key(traj_states, traj_states + states_in_window_size,
							thrust::make_zip_iterator(traj_times, diffs.begin()));

		std::cout << "sort" << std::endl;

		thrust::device_vector<size_t> d_res_states(states_in_window_size);
		thrust::device_vector<float> d_res_times(states_in_window_size);

		auto time_slices_begin =
			thrust::make_transform_iterator(thrust::make_zip_iterator(traj_times, diffs.begin()),
											[w_b, w_e] __host__ __device__(const thrust::tuple<float, float>& t) {
												const float b = thrust::get<0>(t) - thrust::get<1>(t);
												const float e = thrust::get<0>(t);
												return fminf(w_e, e) - fmaxf(w_b, b);
											});

		auto res_end = thrust::reduce_by_key(traj_states, traj_states + states_in_window_size, time_slices_begin,
											 d_res_states.begin(), d_res_times.begin());

		
		std::cout << "reduce" << std::endl;

		size_t res_size = res_end.first - d_res_states.begin();

		std::vector<size_t> states(res_size);
		std::vector<float> times(res_size);

		CUDA_CHECK(hipMemcpy(states.data(), thrust::raw_pointer_cast(d_res_states.data()), res_size * sizeof(size_t),
							  hipMemcpyDeviceToHost));
		CUDA_CHECK(hipMemcpy(times.data(), thrust::raw_pointer_cast(d_res_times.data()), res_size * sizeof(float),
							  hipMemcpyDeviceToHost));

		std::cout << "cpy" << std::endl;

		for (size_t i = 0; i < res_size; ++i)
		{
			probs[window_idx][states[i]] += times[i];
		}

		std::cout << "host upd" << std::endl;
	}
}

int main()
{
	CUDA_CHECK(hipSetDevice(0));

	int trajectories = 1'000'000;
	size_t max_traj_len = 100;

	float max_time = 50.f;
	float window_size = 0.2f;

	size_t* d_states;
	float* d_times;
	hiprandState* d_rands;

	size_t* d_traj_states;
	float* d_traj_times;
	size_t* d_traj_lengths;

	std::vector<std::map<size_t, float>> probs;
	probs.resize(max_time / window_size);

	CUDA_CHECK(hipMalloc(&d_states, trajectories * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_times, trajectories * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_rands, trajectories * sizeof(hiprandState)));

	CUDA_CHECK(hipMalloc(&d_traj_states, trajectories * max_traj_len * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_traj_times, trajectories * max_traj_len * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_traj_lengths, trajectories * max_traj_len * sizeof(size_t)));

	size_t* d_res_states;
	float* d_res_times;

	CUDA_CHECK(hipMalloc(&d_res_states, trajectories * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_res_times, trajectories * sizeof(float)));

	run_initialize(trajectories, 1234, d_states, d_times, d_rands);

	while (trajectories)
	{
		run_simulate(max_time, trajectories, d_states, d_times, d_rands, d_traj_states, d_traj_times, max_traj_len,
					 d_traj_lengths);

		std::cout << "sim" << std::endl;

		statistics_windows_probs(probs, window_size, max_time, thrust::device_pointer_cast(d_traj_states),
								 thrust::device_pointer_cast(d_traj_times), max_traj_len, trajectories);

		CUDA_CHECK(hipMemset(d_traj_times, 0, trajectories * max_traj_len * sizeof(float)));

		auto remaining_traj_begin = thrust::make_zip_iterator(d_states, d_times, d_rands);
		auto remaining_traj_end =
			thrust::make_zip_iterator(d_states + trajectories, d_times + trajectories, d_rands + trajectories);

		remaining_traj_end = thrust::partition(thrust::device, remaining_traj_begin, remaining_traj_end, d_traj_lengths,
											   [max_traj_len] __device__(size_t l) { return l == max_traj_len; });

		trajectories = remaining_traj_end.get_iterator_tuple().get<0>() - d_states;

		std::cout << "one sim " << trajectories << std::endl;

		for (size_t i = 0; i < probs.size(); ++i)
		{
			std::cout << "window " << i << std::endl;
			for (auto& [state, time] : probs[i])
			{
				std::cout << state << " " << time / (1'000'000 * window_size) << std::endl;
			}
		}
	}

	CUDA_CHECK(hipFree(d_states));
	CUDA_CHECK(hipFree(d_times));
	CUDA_CHECK(hipFree(d_rands));
	CUDA_CHECK(hipFree(d_traj_states));
	CUDA_CHECK(hipFree(d_traj_times));
	CUDA_CHECK(hipFree(d_traj_lengths));
	CUDA_CHECK(hipFree(d_res_states));
	CUDA_CHECK(hipFree(d_res_times));

	return 0;
}
