#include <hip/hip_runtime.h>
#include <iostream>

#include <thrust/sort.h>

#include "simulation.h"

void cuda_check(hipError_t e, const char* file, int line)
{
	if (e != hipSuccess)
	{
		std::printf("CUDA API failed at %s:%d with error: %s (%d)\n", file, line, hipGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

#define CUDA_CHECK(func) cuda_check(func, __FILE__, __LINE__)

int main()
{
	CUDA_CHECK(hipSetDevice(0));

	int trajectories = 1'000'000;
	size_t max_traj_len = 100;

	float max_time = 100.f;

	size_t* d_states;
	float* d_times;
	hiprandState* d_rands;

	size_t* d_traj_states;
	float* d_traj_times;
	size_t* d_traj_lengths;
	bool* d_finished;

	CUDA_CHECK(hipMalloc(&d_states, trajectories * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_times, trajectories * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_rands, trajectories * sizeof(hiprandState)));

	CUDA_CHECK(hipMalloc(&d_traj_states, trajectories * max_traj_len * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_traj_times, trajectories * max_traj_len * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_traj_lengths, trajectories * max_traj_len * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_finished, sizeof(bool)));

	size_t* d_res_states;
	float* d_res_times;

	CUDA_CHECK(hipMalloc(&d_res_states, trajectories * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_res_times, trajectories * sizeof(float)));

	run_initialize(trajectories, 1234, d_states, d_times, d_rands);

	while (true)
	{
		run_simulate(max_time, trajectories, d_states, d_times, d_rands, d_traj_states, d_traj_times, max_traj_len,
					 d_traj_lengths, d_finished);


		bool finished;
		CUDA_CHECK(hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost));

		thrust::sort_by_key(thrust::device, d_traj_states, d_traj_states + trajectories * max_traj_len, d_traj_times);

		auto end = thrust::reduce_by_key(thrust::device, d_traj_states, d_traj_states + trajectories * max_traj_len,
										 d_traj_times, d_res_states, d_res_times);

        
		std::cout << "one sim " << end.first - d_res_states << std::endl;

		if (finished)
			break;

		finished = true;
		CUDA_CHECK(hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice));
	}

	CUDA_CHECK(hipFree(d_states));
	CUDA_CHECK(hipFree(d_times));
	CUDA_CHECK(hipFree(d_rands));
	CUDA_CHECK(hipFree(d_traj_states));
	CUDA_CHECK(hipFree(d_traj_times));
	CUDA_CHECK(hipFree(d_traj_lengths));
    CUDA_CHECK(hipFree(d_finished));
    CUDA_CHECK(hipFree(d_res_states));
    CUDA_CHECK(hipFree(d_res_times));

	return 0;
}