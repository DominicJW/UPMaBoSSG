#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <map>
#include <vector>

#include <thrust/adjacent_difference.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/zip_function.h>

#include "simulation.h"
#include "timer.h"

struct in_window_functor
{
	float window_begin, window_end;

	in_window_functor(float window_begin, float window_end) : window_begin(window_begin), window_end(window_end) {}

	__device__ bool operator()(float slice_begin, float slice_end)
	{
		return !(slice_end < window_begin || slice_begin >= window_end) && slice_end != 0.f;
	}
};

void cuda_check(hipError_t e, const char* file, int line)
{
	if (e != hipSuccess)
	{
		std::printf("CUDA API failed at %s:%d with error: %s (%d)\n", file, line, hipGetErrorString(e), e);
		std::exit(EXIT_FAILURE);
	}
}

#define CUDA_CHECK(func) cuda_check(func, __FILE__, __LINE__)

void statistics_windows_probs(std::vector<std::map<size_t, float>>& probs, float window_size, float max_time,
							  thrust::device_ptr<size_t> traj_states, thrust::device_ptr<float> traj_times,
							  size_t max_traj_len, size_t n_trajectories)
{
	constexpr size_t batch_size_limit = 50'000'000;
	size_t windows_count = std::ceil(max_time / window_size);

	timer t;
	long long slices_time = 0.f, count_time = 0.f, partition_time = 0.f, sort_time = 0.f, reduce_time = 0.f,
			  update_time = 0.f;

	t.start();

	// since traj_times does not contain time slices, but the timepoints of transitions,
	// we compute a beginning for each timepoint for convenience
	thrust::device_vector<float> traj_time_starts(n_trajectories * max_traj_len);
	thrust::adjacent_difference(traj_times, traj_times + n_trajectories * max_traj_len, traj_time_starts.begin());
	thrust::transform(traj_times, traj_times + n_trajectories * max_traj_len, traj_time_starts.begin(),
					  traj_time_starts.begin(), thrust::minus<float>());

	t.stop();

	slices_time = t.millisecs();

	// begin and end of the whole traj batch
	auto begin = thrust::make_zip_iterator(traj_states, traj_time_starts.begin(), traj_times);
	auto end = begin + n_trajectories * max_traj_len;

	// host and device result arrays
	thrust::device_vector<size_t> d_res_window_idxs;
	thrust::device_vector<size_t> d_res_states;
	thrust::device_vector<float> d_res_times;
	std::vector<size_t> h_res_window_idxs;
	std::vector<size_t> h_res_states;
	std::vector<float> h_res_times;

	t.start();

	std::vector<size_t> windows_sizes;

	// compute the size of each window
	for (size_t window_idx = 0; window_idx < windows_count; window_idx++)
	{
		float w_b = window_idx * window_size;
		float w_e = w_b + window_size;

		// find states in the window by moving them to the front
		auto time_begin = thrust::make_zip_iterator(traj_time_starts.begin(), traj_times);
		windows_sizes.push_back(thrust::count_if(time_begin, time_begin + n_trajectories * max_traj_len,
												 thrust::make_zip_function(in_window_functor(w_b, w_e))));
	}

	t.stop();

	count_time += t.millisecs();

	// we compute offsets for each window data
	{
		auto whole_size = thrust::reduce(windows_sizes.begin(), windows_sizes.end());
		thrust::exclusive_scan(windows_sizes.begin(), windows_sizes.end(), windows_sizes.begin());
		windows_sizes.push_back(whole_size);
	}

	// divide windows to batches of batch_size_limit so we do not OOM
	std::vector<size_t> batch_indices;
	{
		batch_indices.push_back(0);
		size_t batch_idx_begin = 0;
		for (int i = 0; i < windows_sizes.size(); i++)
		{
			if ((windows_sizes[i] - windows_sizes[batch_idx_begin]) < batch_size_limit && i != windows_sizes.size() - 1)
				continue;

			batch_indices.push_back(i);
			batch_idx_begin = i;
		}
	}

	thrust::device_vector<size_t> batch_states;
	thrust::device_vector<int> batch_window_idxs;
	thrust::device_vector<float> batch_time_starts, batch_time_ends;

	// we compute a batch of windows at a time
	for (int i = 0; i < batch_indices.size() - 1; i++)
	{
		size_t batch_idx_begin = batch_indices[i];
		size_t batch_idx_end = batch_indices[i + 1];
		size_t batch_size = windows_sizes[batch_idx_end] - windows_sizes[batch_idx_begin];

		batch_states.resize(batch_size);
		batch_window_idxs.resize(batch_size);
		batch_time_starts.resize(batch_size);
		batch_time_ends.resize(batch_size);

		auto batch_begin =
			thrust::make_zip_iterator(batch_states.begin(), batch_time_starts.begin(), batch_time_ends.begin());

		t.start();

		// we fill in batch arrays with windows in this batch
		for (size_t window_idx = batch_idx_begin; window_idx < batch_idx_end; window_idx++)
		{
			size_t in_batch_offset = windows_sizes[window_idx] - windows_sizes[batch_idx_begin];
			float w_b = window_idx * window_size;
			float w_e = w_b + window_size;

			auto key_begin = thrust::make_zip_iterator(traj_time_starts.begin(), traj_times);
			thrust::copy_if(begin, begin + n_trajectories * max_traj_len, key_begin, batch_begin + in_batch_offset,
							thrust::make_zip_function(in_window_functor(w_b, w_e)));

			thrust::fill(batch_window_idxs.begin() + in_batch_offset,
						 batch_window_idxs.begin() + windows_sizes[window_idx + 1] - windows_sizes[batch_idx_begin],
						 window_idx);
		}

		t.stop();

		partition_time += t.millisecs();

		t.start();

		// let us sort ((window_idx, state, time_b, time_e)) array by (window_idx, state) key
		// so we can reduce it in the next step
		auto key_begin = thrust::make_zip_iterator(batch_window_idxs.begin(), batch_states.begin());
		auto data_begin = thrust::make_zip_iterator(batch_time_starts.begin(), batch_time_ends.begin());
		thrust::sort_by_key(key_begin, key_begin + batch_size, data_begin);

		t.stop();

		sort_time += t.millisecs();

		// create transform iterator, which computes the intersection of a window and a transition time slice
		auto time_slices_begin = thrust::make_transform_iterator(
			thrust::make_zip_iterator(batch_time_starts.begin(), batch_time_ends.begin(), batch_window_idxs.begin()),
			[window_size, max_time] __host__ __device__(const thrust::tuple<float, float, size_t>& t) {
				const float b = thrust::get<0>(t);
				const float e = thrust::get<1>(t);

				const float w_b = thrust::get<2>(t) * window_size;
				const float w_e = fminf(w_b + window_size, max_time);

				return fminf(w_e, e) - fmaxf(w_b, b);
			});

		t.start();

		// we compute the size of the result (sum of unique states in each window)
		size_t result_size = thrust::unique_count(key_begin, key_begin + batch_size);

		d_res_window_idxs.resize(result_size);
		d_res_states.resize(result_size);
		d_res_times.resize(result_size);

		// reduce sorted array of (state, time_slice)
		// after this we have unique states in the first result array and sum of slices in the second result array
		thrust::reduce_by_key(key_begin, key_begin + batch_size, time_slices_begin,
							  thrust::make_zip_iterator(d_res_window_idxs.begin(), d_res_states.begin()),
							  d_res_times.begin());

		t.stop();

		reduce_time += t.millisecs();

		t.start();

		h_res_window_idxs.resize(result_size);
		h_res_states.resize(result_size);
		h_res_times.resize(result_size);

		// copy result data into host
		CUDA_CHECK(hipMemcpy(h_res_window_idxs.data(), thrust::raw_pointer_cast(d_res_window_idxs.data()),
							  result_size * sizeof(size_t), hipMemcpyDeviceToHost));
		CUDA_CHECK(hipMemcpy(h_res_states.data(), thrust::raw_pointer_cast(d_res_states.data()),
							  result_size * sizeof(size_t), hipMemcpyDeviceToHost));
		CUDA_CHECK(hipMemcpy(h_res_times.data(), thrust::raw_pointer_cast(d_res_times.data()),
							  result_size * sizeof(float), hipMemcpyDeviceToHost));

		// update
		for (size_t i = 0; i < result_size; ++i)
		{
			probs[h_res_window_idxs[i]][h_res_states[i]] += h_res_times[i];
		}

		t.stop();

		update_time += t.millisecs();

		batch_idx_begin = batch_idx_end;
	}

	// print diagnostics
	std::cout << "slices_time: " << slices_time << "ms" << std::endl;
	std::cout << "partition_time: " << partition_time << "ms" << std::endl;
	std::cout << "sort_time: " << sort_time << "ms" << std::endl;
	std::cout << "reduce_time: " << reduce_time << "ms" << std::endl;
	std::cout << "update_time: " << update_time << "ms" << std::endl;
}

int main()
{
	CUDA_CHECK(hipSetDevice(0));

	const int o_trajectories = 1'000'000;
	int trajectories = o_trajectories;
	size_t max_traj_len = 100;

	float max_time = 5.f;
	float window_size = 0.2f;

	size_t* d_states;
	float* d_times;
	hiprandState* d_rands;

	size_t* d_traj_states;
	float* d_traj_times;
	size_t* d_traj_lengths;

	std::vector<std::map<size_t, float>> probs;
	probs.resize((size_t)std::ceil(max_time / window_size));

	CUDA_CHECK(hipMalloc(&d_states, trajectories * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_times, trajectories * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_rands, trajectories * sizeof(hiprandState)));

	CUDA_CHECK(hipMalloc(&d_traj_states, trajectories * max_traj_len * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_traj_times, trajectories * max_traj_len * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_traj_lengths, trajectories * max_traj_len * sizeof(size_t)));

	size_t* d_res_states;
	float* d_res_times;

	CUDA_CHECK(hipMalloc(&d_res_states, trajectories * sizeof(size_t)));
	CUDA_CHECK(hipMalloc(&d_res_times, trajectories * sizeof(float)));

	run_initialize(trajectories, 1234, d_states, d_times, d_rands);

	while (trajectories)
	{
		run_simulate(max_time, trajectories, d_states, d_times, d_rands, d_traj_states, d_traj_times, max_traj_len,
					 d_traj_lengths);

		std::cout << "sim" << std::endl;

		statistics_windows_probs(probs, window_size, max_time, thrust::device_pointer_cast(d_traj_states),
								 thrust::device_pointer_cast(d_traj_times), max_traj_len, trajectories);

		CUDA_CHECK(hipMemset(d_traj_times, 0, trajectories * max_traj_len * sizeof(float)));

		auto remaining_traj_begin = thrust::make_zip_iterator(d_states, d_times, d_rands);
		auto remaining_traj_end =
			thrust::make_zip_iterator(d_states + trajectories, d_times + trajectories, d_rands + trajectories);

		remaining_traj_end = thrust::partition(thrust::device, remaining_traj_begin, remaining_traj_end, d_traj_lengths,
											   [max_traj_len] __device__(size_t l) { return l == max_traj_len; });

		trajectories = remaining_traj_end.get_iterator_tuple().get<0>() - d_states;

		// std::cout << "one sim " << trajectories << std::endl;

		for (size_t i = 0; i < probs.size(); ++i)
		{
			std::cout << "window " << i << std::endl;
			for (auto& [state, time] : probs[i])
			{
				std::cout << state << " " << time / (o_trajectories * window_size) << std::endl;
			}
		}
	}

	CUDA_CHECK(hipFree(d_states));
	CUDA_CHECK(hipFree(d_times));
	CUDA_CHECK(hipFree(d_rands));
	CUDA_CHECK(hipFree(d_traj_states));
	CUDA_CHECK(hipFree(d_traj_times));
	CUDA_CHECK(hipFree(d_traj_lengths));
	CUDA_CHECK(hipFree(d_res_states));
	CUDA_CHECK(hipFree(d_res_times));

	return 0;
}
