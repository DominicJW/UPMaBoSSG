#include "hip/hip_runtime.h"
#include <>

#include <thrust/device_free.h>
#include <thrust/device_malloc.h>

#include "../diagnostics.h"
#include "../utils.h"
#include "window_average_small.h"
#include <fstream>
__device__ int get_non_internal_index(const state_t& s, const state_t& internal_mask)
{
	int idx = 0;
	int idx_i = 0;

	for (int i = 0; i < states_count; i++)
	{
		if (!internal_mask.is_set(i))
		{
			int multiplier = s.is_set(i) ? 1 : 0;

			idx += multiplier * (1 << idx_i);

			idx_i++;
		}
	}

	return idx;
}

__global__ void window_average_small(int max_traj_len, int n_trajectories, float window_size, state_t internal_mask,
									 int noninternal_states_count, const state_t* __restrict__ traj_states,
									 const float* __restrict__ traj_times, const float* __restrict__ traj_tr_entropies,
									 float* __restrict__ window_probs, float* __restrict__ window_tr_entropies)
{
	auto id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= n_trajectories * max_traj_len)
		return;

	if (id % max_traj_len == 0 || traj_times[id] == 0.f)
	{
		return;
	}

	const int state_idx = get_non_internal_index(traj_states[id], internal_mask);
	const float tr_h = traj_tr_entropies[id];

	float slice_begin = traj_times[id - 1];
	float slice_end = traj_times[id];
	int wnd_idx = floorf(slice_begin / window_size);

	while (slice_end > slice_begin)
	{
		float wnd_end = (wnd_idx + 1) * window_size;
		float slice_in_wnd = fminf(slice_end, wnd_end) - slice_begin;

		atomicAdd(window_probs + (wnd_idx * noninternal_states_count + state_idx), slice_in_wnd);
		atomicAdd(window_tr_entropies + wnd_idx, tr_h * slice_in_wnd);

		wnd_idx++;
		slice_begin = fminf(slice_end, wnd_end);
	}
}

__global__ void window_average_small_discrete(int max_traj_len, int n_trajectories, float window_size,
											  state_t internal_mask, int noninternal_states_count,
											  const state_t* __restrict__ traj_states,
											  const float* __restrict__ traj_times,
											  const float* __restrict__ traj_tr_entropies,
											  int* __restrict__ window_probs, float* __restrict__ window_tr_entropies)
{
	auto id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id >= n_trajectories * max_traj_len)
		return;

	if (id % max_traj_len == 0 || traj_times[id] == 0.f)
	{
		return;
	}

	const int state_idx = get_non_internal_index(traj_states[id], internal_mask);
	const float tr_h = traj_tr_entropies[id];

	int wnd_idx = lroundf(traj_times[id - 1] / window_size);

	atomicAdd(window_probs + (wnd_idx * noninternal_states_count + state_idx), 1);
	atomicAdd(window_tr_entropies + wnd_idx, tr_h);
}

window_average_small_stats::window_average_small_stats(float window_size, float max_time, bool discrete_time,
													   state_t internal_mask, size_t non_internals, size_t max_traj_len,
													   size_t max_n_trajectories)
	: window_size_(window_size),
	  max_time_(max_time),
	  discrete_time_(discrete_time),
	  internal_mask_(internal_mask),
	  noninternal_states_count_(1 << non_internals),
	  max_traj_len_(max_traj_len),
	  max_n_trajectories_(max_n_trajectories)
{
	timer t;
	t.start();

	size_t windows_count = std::ceil(max_time / window_size);

	window_tr_entropies_ = thrust::device_malloc<float>(windows_count);
	result_tr_entropies_.resize(windows_count);
	CUDA_CHECK(hipMemset(window_tr_entropies_.get(), 0, windows_count * sizeof(float)));

	if (discrete_time)
	{
		window_probs_discrete_ = thrust::device_malloc<int>(windows_count * noninternal_states_count_);
		result_probs_discrete_.resize(windows_count * noninternal_states_count_);
		CUDA_CHECK(
			hipMemset(window_probs_discrete_.get(), 0, windows_count * noninternal_states_count_ * sizeof(int)));
	}
	else
	{
		window_probs_ = thrust::device_malloc<float>(windows_count * noninternal_states_count_);
		result_probs_.resize(windows_count * noninternal_states_count_);
		CUDA_CHECK(hipMemset(window_probs_.get(), 0, windows_count * noninternal_states_count_ * sizeof(float)));
	}

	CUDA_CHECK(hipDeviceSynchronize());

	t.stop();

	if (print_diags)
	{
		std::cout << "window_average_small> init_time: " << t.millisecs() << "ms" << std::endl;
	}
}

window_average_small_stats::~window_average_small_stats()
{
	thrust::device_free(window_probs_);
	thrust::device_free(window_tr_entropies_);
}

void window_average_small_stats::process_batch(thrust::device_ptr<state_t> traj_states,
											   thrust::device_ptr<float> traj_times,
											   thrust::device_ptr<float> traj_tr_entropies, thrust::device_ptr<state_t>,
											   thrust::device_ptr<trajectory_status>, int n_trajectories)
{
	process_batch_internal(traj_states, traj_times, traj_tr_entropies, n_trajectories);
}

void window_average_small_stats::process_batch_internal(thrust::device_ptr<state_t> traj_states,
														thrust::device_ptr<float> traj_times,
														thrust::device_ptr<float> traj_tr_entropies, int n_trajectories)
{
	timer t;
	t.start();

	if (discrete_time_)
	{
		window_average_small_discrete<<<DIV_UP(max_traj_len_ * n_trajectories, 512), 512>>>(
			max_traj_len_, n_trajectories, window_size_, internal_mask_, noninternal_states_count_, traj_states.get(),
			traj_times.get(), traj_tr_entropies.get(), window_probs_discrete_.get(), window_tr_entropies_.get());
	}
	else
	{
		window_average_small<<<DIV_UP(max_traj_len_ * n_trajectories, 512), 512>>>(
			max_traj_len_, n_trajectories, window_size_, internal_mask_, noninternal_states_count_, traj_states.get(),
			traj_times.get(), traj_tr_entropies.get(), window_probs_.get(), window_tr_entropies_.get());
	}

	CUDA_CHECK(hipDeviceSynchronize());

	t.stop();

	if (print_diags)
	{
		std::cout << "window_average_small> reduce_time: " << t.millisecs() << "ms" << std::endl;
	}
}

void window_average_small_stats::finalize()
{
	size_t windows_count = std::ceil(max_time_ / window_size_);
	timer t;

	t.start();

	// copy result data into host
	if (discrete_time_)
		CUDA_CHECK(hipMemcpy(result_probs_discrete_.data(), thrust::raw_pointer_cast(window_probs_discrete_),
							  windows_count * noninternal_states_count_ * sizeof(int), hipMemcpyDeviceToHost));
	else
		CUDA_CHECK(hipMemcpy(result_probs_.data(), thrust::raw_pointer_cast(window_probs_),
							  windows_count * noninternal_states_count_ * sizeof(float), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(result_tr_entropies_.data(), thrust::raw_pointer_cast(window_tr_entropies_),
						  windows_count * sizeof(float), hipMemcpyDeviceToHost));

	t.stop();

	if (print_diags)
	{
		std::cout << "window_average_small> finalize_time: " << t.millisecs() << "ms" << std::endl;
	}
}

state_t non_internal_idx_to_state(const state_t& internal_mask, int idx)
{
	state_t ret;
	size_t idx_i = 0;
	for (size_t i = 0; i < states_count; i++)
	{
		if (!internal_mask.is_set(i))
		{
			if ((idx & (1 << idx_i)) != 0)
				ret.set(i);
			idx_i++;
		}
	}

	return ret;
}

float window_average_small_stats::get_single_result_prob(int n_trajectories, size_t idx)
{
	if (discrete_time_)
	{
		auto occurences = result_probs_discrete_[idx];

		return (float)occurences / (float)n_trajectories;
	}
	else
	{
		auto cumul_slices = result_probs_[idx];

		return cumul_slices / (n_trajectories * window_size_);
	}
}

void window_average_small_stats::visualize(int n_trajectories, const std::vector<std::string>& nodes)
{
	size_t windows_count = std::ceil(max_time_ / window_size_);

	for (size_t i = 0; i < windows_count; ++i)
	{
		float entropy = 0.f;
		float wnd_tr_entropy = result_tr_entropies_[i] / n_trajectories;
		wnd_tr_entropy /= discrete_time_ ? 1 : window_size_;

		for (size_t s_idx = 0; s_idx < noninternal_states_count_; s_idx++)
		{
			auto prob = get_single_result_prob(n_trajectories, i * noninternal_states_count_ + s_idx);

			if (prob == 0.f)
				continue;

			entropy += -std::log2(prob) * prob;
		}

		std::cout << "window (" << i * window_size_ << ", " << (i + 1) * window_size_ << "]" << std::endl;
		std::cout << "entropy: " << entropy << std::endl;
		std::cout << "transition entropy: " << wnd_tr_entropy << std::endl;

		for (size_t s_idx = 0; s_idx < noninternal_states_count_; s_idx++)
		{
			auto prob = get_single_result_prob(n_trajectories, i * noninternal_states_count_ + s_idx);

			if (prob == 0.f)
				continue;

			std::cout << prob << " " << to_string(non_internal_idx_to_state(internal_mask_, s_idx), nodes) << std::endl;
		}
	}
}


void window_average_small_stats::writeCSV(int n_trajectories, const std::vector<std::string>& nodes, const std::string prefix)
{
	size_t windows_count = std::ceil(max_time_ / window_size_);
	std::ofstream ofs;
	
	ofs.open(prefix + "_probtraj.csv");
	if (ofs) 
	{
		// Computing max states for header
		int max_states = 0;
		for (size_t i = 0; i < windows_count; ++i)
		{
			int num_states = 0;			
			for (size_t s_idx = 0; s_idx < noninternal_states_count_; s_idx++)
			{
				auto prob = get_single_result_prob(n_trajectories, i * noninternal_states_count_ + s_idx);

				if (prob == 0.f)
					continue;
					
				num_states += 1;
			}
			
			max_states = std::max(max_states, num_states);
		}
		
		// writing header
		ofs << "Time\tTH\tErrorTH\tH\tHD=0\t";
		for (int i = 0; i < max_states; i++)
		{
			ofs << "State\tProba\tErrorProba";
			if (i < max_states - 1) {
				ofs << "\t";
			}
		}
		
		for (size_t i = 0; i < windows_count; ++i)
		{
			float entropy = 0.f;
			float wnd_tr_entropy = result_tr_entropies_[i] / n_trajectories;
			wnd_tr_entropy /= discrete_time_ ? 1 : window_size_;

			for (size_t s_idx = 0; s_idx < noninternal_states_count_; s_idx++)
			{
				auto prob = get_single_result_prob(n_trajectories, i * noninternal_states_count_ + s_idx);

				if (prob == 0.f)
					continue;

				entropy += -std::log2(prob) * prob;
			}
			ofs << i * window_size_ << "\t";
			// std::cout << "window (" << i * window_size_ << ", " << (i + 1) * window_size_ << "]" << std::endl;
			// std::cout << "entropy: " << entropy << std::endl;
			// std::cout << "transition entropy: " << wnd_tr_entropy << std::endl;
			ofs << wnd_tr_entropy << "\t" << 0.f << "\t" << entropy << "\t" << 0.f << "\t";

			for (size_t s_idx = 0; s_idx < noninternal_states_count_; s_idx++)
			{
				auto prob = get_single_result_prob(n_trajectories, i * noninternal_states_count_ + s_idx);

				if (prob == 0.f)
					continue;

				std::cout << prob << " " << to_string(non_internal_idx_to_state(internal_mask_, s_idx), nodes) << std::endl;
				ofs << to_string(non_internal_idx_to_state(internal_mask_, s_idx), nodes) << "\t" << prob << "\t" << 0.f;
				if (s_idx < (noninternal_states_count_ - 1))
				{
					ofs << "\t";
				}
			}
			ofs << std::endl;
		}
	}
}
