#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "cfg_config.h.generated"
#include "simulation.h"
#include "transition_rates.cu.generated"

#define DIV_UP(x, y) (x + y - 1) / y

__device__ void compute_transition_rates(float* __restrict__ transition_rates, const state_t& state);

__device__ int select_flip_bit(const float* __restrict__ transition_rates, float total_rate,
							   hiprandState* __restrict__ rand)
{
	float r = hiprand_uniform(rand) * total_rate;
	float sum = 0;
	for (int i = 0; i < states_count; i++)
	{
		sum += transition_rates[i];
		if (r < sum)
			return i;
	}
	return states_count - 1;
}

__global__ void initialize(int trajectories_count, unsigned long long seed, state_t fixed_part, state_t free_mask,
						   state_t* __restrict__ states, float* __restrict__ times, hiprandState* __restrict__ rands)
{
	auto id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= trajectories_count)
		return;

	// initialize random number generator
	hiprand_init(seed, id, 0, rands + id);

	// initialize state
	state_t s = fixed_part;
	{
		// randomly set free vars
		for (int i = 0; i < states_count; i++)
		{
			if (free_mask.is_set(i) && hiprand_uniform(rands + id) > 0.5f)
				s.set(i);
		}
	}
	states[id] = s;

	// printf("state %i\n", (int)states[id].data[0]);

	// set time to zero
	times[id] = 0.f;
}

void run_initialize(int trajectories_count, unsigned long long seed, state_t fixed_part, state_t free_mask,
					state_t* states, float* times, hiprandState* rands)
{
	initialize<<<DIV_UP(trajectories_count, 256), 256>>>(trajectories_count, seed, fixed_part, free_mask, states, times,
														 rands);
}

template <bool discrete_time>
__global__ void simulate(float max_time, float time_tick, int trajectories_count, int trajectory_limit,
						 state_t* __restrict__ last_states, float* __restrict__ last_times,
						 hiprandState* __restrict__ rands, state_t* __restrict__ trajectory_states,
						 float* __restrict__ trajectory_times, trajectory_status* __restrict__ trajectory_statuses)
{
	auto id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= trajectories_count)
		return;

	float transition_rates[states_count];

	// Initialize thread variables
	hiprandState rand = rands[id];
	state_t state = last_states[id];
	float time = last_times[id];
	int step = 0;
	trajectory_states = trajectory_states + id * trajectory_limit;
	trajectory_times = trajectory_times + id * trajectory_limit;
	trajectory_status status = trajectory_status::CONTINUE;

	// as the first time set the last from the prev run
	trajectory_times[step++] = time;

	while (true)
	{
		// get transition rates for current state
		compute_transition_rates(transition_rates, state);

		// sum up transition rates
		float total_rate = 0.f;
		for (size_t i = 0; i < states_count; i++)
			total_rate += transition_rates[i];

		// if total rate is zero, no transition is possible
		if (total_rate == 0.f)
		{
			status = trajectory_status::FIXED_POINT;
			time = max_time;
		}
		else
		{
			if constexpr (discrete_time)
				time += time_tick;
			else
				time += -logf(hiprand_uniform(&rand)) / total_rate;
		}

		trajectory_states[step] = state;
		trajectory_times[step] = time;
		step++;

		if (time >= max_time || step >= trajectory_limit)
			break;

		int flip_bit = select_flip_bit(transition_rates, total_rate, &rand);
		state.flip(flip_bit);

		// printf("thread %i flip bit %i next state %i\n", id, flip_bit, state);
	}

	// save thread variables
	rands[id] = rand;
	last_states[id] = state;
	last_times[id] = time;

	if (status != trajectory_status::FIXED_POINT)
	{
		status = (time >= max_time) ? trajectory_status::FINISHED : trajectory_status::CONTINUE;
	}

	trajectory_statuses[id] = status;
}

void run_simulate(float max_time, float time_tick, bool discrete_time, int trajectories_count, int trajectory_limit,
				  state_t* last_states, float* last_times, hiprandState* rands, state_t* trajectory_states,
				  float* trajectory_times, trajectory_status* trajectory_statuses)
{
	if (discrete_time)
		simulate<true><<<DIV_UP(trajectories_count, 256), 256>>>(
			max_time, time_tick, trajectories_count, trajectory_limit, last_states, last_times, rands,
			trajectory_states, trajectory_times, trajectory_statuses);
	else
		simulate<false><<<DIV_UP(trajectories_count, 256), 256>>>(
			max_time, time_tick, trajectories_count, trajectory_limit, last_states, last_times, rands,
			trajectory_states, trajectory_times, trajectory_statuses);
}
