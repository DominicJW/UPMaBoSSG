#include "hip/hip_runtime.h"
#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/partition.h>

#include "diagnostics.h"
#include "simulation.h"
#include "simulation_runner.h"
#include "utils.h"


template <typename T>
struct eq_ftor
{
	T it;

	eq_ftor(T it) : it(it) {}

	__device__ bool operator()(T other) { return other == it; }
};

simulation_runner::simulation_runner(int n_trajectories, seed_t seed, state_t fixed_initial_part, state_t free_mask,
									 float max_time, float time_tick, bool discrete_time, state_t internal_mask,
									 std::vector<float> variables_values)
	: n_trajectories_(n_trajectories),
	  seed_(seed),
	  max_time_(max_time),
	  time_tick_(time_tick),
	  discrete_time_(discrete_time),
	  fixed_initial_part_(fixed_initial_part),
	  free_mask_(free_mask),
	  variables_values_(std::move(variables_values))
{
	trajectory_batch_limit = std::min(1'000'000, n_trajectories);
	trajectory_len_limit = 100; // TODO compute limit according to the available mem

	for (int i = 0; i < states_count; i++)
	{
		if (!internal_mask.is_set(i))
			noninternal_indices_.push_back(i);
	}
}

void simulation_runner::run_simulation(stats_composite& stats_runner)
{
	timer t;
	long long init_time = 0.f, simulation_time = 0.f, preparation_time = 0.f, stats_time = 0.f;
	int remaining_trajs = n_trajectories_;

	t.start();

	CUDA_CHECK(hipSetDevice(0));

	auto d_last_states = thrust::device_malloc<state_t>(trajectory_batch_limit);
	auto d_last_times = thrust::device_malloc<float>(trajectory_batch_limit);
	auto d_rands = thrust::device_malloc<hiprandState>(trajectory_batch_limit);

	auto d_traj_states = thrust::device_malloc<state_t>(trajectory_batch_limit * trajectory_len_limit);
	auto d_traj_times = thrust::device_malloc<float>(trajectory_batch_limit * trajectory_len_limit);
	auto d_traj_tr_entropies = thrust::device_malloc<float>(trajectory_batch_limit * trajectory_len_limit);
	auto d_traj_statuses = thrust::device_malloc<trajectory_status>(trajectory_batch_limit);

	// initialize states
	run_initialize_random(trajectory_batch_limit, seed_, d_rands.get());

	run_initialize_initial_state(trajectory_batch_limit, fixed_initial_part_, free_mask_, d_last_states.get(),
								 d_last_times.get(), d_rands.get());

	set_boolean_function_variable_values(variables_values_.data());
	set_noninternal_indices(noninternal_indices_.data(), noninternal_indices_.size());

	CUDA_CHECK(hipMemset(d_traj_times.get(), 0, trajectory_batch_limit * trajectory_len_limit * sizeof(float)));

	CUDA_CHECK(hipDeviceSynchronize());

	t.stop();

	init_time = t.millisecs();

	int trajectories_in_batch = std::min(n_trajectories_, trajectory_batch_limit);
	n_trajectories_ -= trajectories_in_batch;

	while (trajectories_in_batch)
	{
		t.start();

		// run single simulation
		run_simulate(max_time_, time_tick_, discrete_time_, (int)noninternal_indices_.size(), trajectories_in_batch,
					 trajectory_len_limit, d_last_states.get(), d_last_times.get(), d_rands.get(), d_traj_states.get(),
					 d_traj_times.get(), d_traj_tr_entropies.get(), d_traj_statuses.get());

		CUDA_CHECK(hipDeviceSynchronize());
		
		t.stop();
		simulation_time += t.millisecs();

		t.start();

		// compute statistics over the simulated trajs
		stats_runner.process_batch(d_traj_states, d_traj_times, d_traj_tr_entropies, d_last_states, d_traj_statuses,
								   trajectories_in_batch);

		t.stop();
		stats_time += t.millisecs();

		// prepare for the next iteration
		{
			t.start();

			// move unfinished trajs to the front and update trajectories_in_batch
			{
				auto thread_state_begin = thrust::make_zip_iterator(d_last_states, d_last_times, d_rands);
				auto remaining_trajectories_in_batch =
					thrust::partition(thread_state_begin, thread_state_begin + trajectories_in_batch, d_traj_statuses,
									  eq_ftor<trajectory_status>(trajectory_status::CONTINUE))
					- thread_state_begin;

				remaining_trajs -= trajectories_in_batch - remaining_trajectories_in_batch;
				trajectories_in_batch = remaining_trajectories_in_batch;
			}

			// add new work to the batch
			{
				int batch_free_size = trajectory_batch_limit - trajectories_in_batch;
				int new_batch_addition = std::min(batch_free_size, n_trajectories_);

				if (new_batch_addition)
				{
					run_initialize_initial_state(new_batch_addition, fixed_initial_part_, free_mask_,
												 d_last_states.get() + trajectories_in_batch,
												 d_last_times.get() + trajectories_in_batch,
												 d_rands.get() + trajectories_in_batch);


					trajectories_in_batch += new_batch_addition;
					n_trajectories_ -= new_batch_addition;
				}
			}

			// set all batch traj times to 0
			CUDA_CHECK(hipMemset(d_traj_times.get(), 0, trajectories_in_batch * trajectory_len_limit * sizeof(float)));

			CUDA_CHECK(hipDeviceSynchronize());

			t.stop();
			preparation_time += t.millisecs();

			if (print_diags)
			{
				std::cout << "simulation_runner> remaining trajs: " << remaining_trajs << std::endl;
			}
		}
	}

	if (print_diags)
	{
		std::cout << "simulation_runner> init_time: " << init_time << "ms" << std::endl;
		std::cout << "simulation_runner> simulation_time: " << simulation_time << "ms" << std::endl;
		std::cout << "simulation_runner> preparation_time: " << preparation_time << "ms" << std::endl;
		std::cout << "simulation_runner> stats_time: " << stats_time << "ms" << std::endl;
	}

	thrust::device_free(d_last_states);
	thrust::device_free(d_last_times);
	thrust::device_free(d_rands);
	thrust::device_free(d_traj_states);
	thrust::device_free(d_traj_times);
	thrust::device_free(d_traj_tr_entropies);
	thrust::device_free(d_traj_statuses);
}
