#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/partition.h>

#include "simulation.h"
#include "simulation_runner.h"
#include "timer.h"
#include "utils.h"

constexpr bool print_diags = false;

template <typename T>
struct eq_ftor
{
	T it;

	eq_ftor(T it) : it(it) {}

	__device__ bool operator()(T other) { return other == it; }
};

simulation_runner::simulation_runner(int n_trajectories, seed_t seed, state_t fixed_initial_part, state_t free_mask,
									 float max_time, float time_tick, bool discrete_time)
	: n_trajectories_(n_trajectories),
	  seed_(seed),
	  max_time_(max_time),
	  time_tick_(time_tick),
	  discrete_time_(discrete_time),
	  fixed_initial_part_(fixed_initial_part),
	  free_mask_(free_mask)
{
	trajectory_len_limit_ = 100; // TODO compute limit according to the available mem
}

void simulation_runner::run_simulation(statistics_func_t run_statistics)
{
	CUDA_CHECK(hipSetDevice(0));

	auto d_last_states = thrust::device_malloc<state_t>(n_trajectories_);
	auto d_last_times = thrust::device_malloc<float>(n_trajectories_);
	auto d_rands = thrust::device_malloc<hiprandState>(n_trajectories_);

	auto d_traj_states = thrust::device_malloc<state_t>(n_trajectories_ * trajectory_len_limit_);
	auto d_traj_times = thrust::device_malloc<float>(n_trajectories_ * trajectory_len_limit_);
	auto d_traj_tr_entropies = thrust::device_malloc<float>(n_trajectories_ * trajectory_len_limit_);
	auto d_traj_statuses = thrust::device_malloc<trajectory_status>(n_trajectories_);

	// initialize states
	run_initialize(n_trajectories_, seed_, fixed_initial_part_, free_mask_, d_last_states.get(), d_last_times.get(),
				   d_rands.get());

	CUDA_CHECK(hipMemset(d_traj_times.get(), 0, n_trajectories_ * trajectory_len_limit_ * sizeof(float)));

	timer t;
	long long simulation_time = 0.f, preparation_time = 0.f;

	while (n_trajectories_)
	{
		t.start();

		// run single simulation
		run_simulate(max_time_, time_tick_, discrete_time_, n_trajectories_, trajectory_len_limit_, d_last_states.get(),
					 d_last_times.get(), d_rands.get(), d_traj_states.get(), d_traj_times.get(),
					 d_traj_tr_entropies.get(), d_traj_statuses.get());

		CUDA_CHECK(hipDeviceSynchronize());

		t.stop();
		simulation_time += t.millisecs();

		// compute statistics over the simulated trajs
		run_statistics(d_traj_states, d_traj_times, d_traj_tr_entropies, d_last_states, d_traj_statuses,
					   trajectory_len_limit_, n_trajectories_);

		// prepare for the next iteration
		{
			t.start();

			// set all traj times to 0
			CUDA_CHECK(hipMemset(d_traj_times.get(), 0, n_trajectories_ * trajectory_len_limit_ * sizeof(float)));

			// move unfinished trajs to the front
			// update n_trajectories_
			auto thread_state_begin = thrust::make_zip_iterator(d_last_states, d_last_times, d_rands);
			n_trajectories_ =
				thrust::partition(thread_state_begin, thread_state_begin + n_trajectories_, d_traj_statuses,
								  eq_ftor<trajectory_status>(trajectory_status::CONTINUE))
							  - thread_state_begin;

			t.stop();
			preparation_time += t.millisecs();

			if (print_diags)
			{
				std::cout << "simulation_runner> remaining trajs: " << n_trajectories_ << std::endl;
			}
		}
	}

	if (print_diags)
	{
		std::cout << "simulation_runner> simulation_time: " << simulation_time << "ms" << std::endl;
		std::cout << "simulation_runner> preparation_time: " << preparation_time << "ms" << std::endl;
	}

	thrust::device_free(d_last_states);
	thrust::device_free(d_last_times);
	thrust::device_free(d_rands);
	thrust::device_free(d_traj_states);
	thrust::device_free(d_traj_times);
	thrust::device_free(d_traj_tr_entropies);
	thrust::device_free(d_traj_statuses);
}
